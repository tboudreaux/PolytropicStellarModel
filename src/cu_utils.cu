#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<thrust/complex.h>

#include "cu_utils.cuh"

typedef double (* odeModel)(double vN, double *argv, int argc);

using namespace std;

__device__ double a(int k, float n){
	// Recursion halting conditions based on boundary conditions
	// that theta(0) = 1 and that thetadot(0) = 0
	if (k == 0){
		return 1;
	}
	else if (k == 1){
		return 0;
	}
	else{
		// relation determined between a, c, and k in part one of project
		return -(c(k-2, n)/(pow((double)k, (double)2)+k));
	}
}


__device__ double c(int m, float n){
	// halting condition from formal power serise definition
	if (m == 0){
		return pow((double)a(0, n), (double)n);
	}
	else{
		double sum = 0;
		for (int k = 1; k <= m; k++){
			// relation from formal power serise
			sum += (k*n-m+k)*a(k, n)*c(m-k, n);
		}
		return (1/(m*a(0, n)))*sum;
	}

}

__device__ double theta_approx(double xi, float n, int itr){
	double theta = 0;
	for (int k=0; k<itr; k++){
		// General form of a power serise
		theta += a(k, n)*pow((double)xi, (double)k);
	}
	return theta;
}

__device__ double vdot_nonDegenerate(double vN, double *argv, int argc){
	return 1;
	// Use complex numbers because base could be negative
	thrust::complex<double> base = argv[1];
	thrust::complex<double> exp = argv[2];
	thrust::complex<double> secondTerm = pow(base, exp);
	return (-2/argv[0])*vN-secondTerm.real();
}

__device__ void single_polytrope(double* xiL, double* state, long int nXi, double polytropicIndex, double* parsedArgv, int polytropeNumber){
	__shared__ double modelArgv[3];
	double* modelState = NULL;
	double h = parsedArgv[1];


	// Pointer to the subpart of the array for this thread
	modelState = state + nXi*2*polytropeNumber;
	for(int i=0; i<nXi; i++){
		if (i==0){
			// Set the initial theta(xi) value based on the power serise expansion
			modelState[i*2] = theta_approx(modelState[i*2], polytropicIndex, 10);
		}	
		else{
			modelArgv[0] = xiL[i];
			modelArgv[1] = modelState[(i-1)*2];
			modelArgv[2] = polytropicIndex;
			// Integrate with rk4
			modelState[i*2+1] = rk4(modelState[(i-1)*2+1], h, (odeModel)vdot_nonDegenerate, modelArgv, 3);
			modelState[i*2] = modelState[i*2+1]*h + modelState[(i-1)*2];
			// When the dimensionless density goes negative constrain it to zero
		}
	}
}

__global__ void distribute_jobs(double* xiL, double* state, long int nXi, int totalModels, int TILELENGTH, double* parsedArgv){
	int polytropeNumber = blockIdx.x*TILELENGTH + threadIdx.x;
	/* printf("%d out of %d\n", polytropeNumber, totalModels); */
	if (polytropeNumber < totalModels)
	{
		// Distribute the polytropic index based on location in Grid
		float polytropicIndex = (polytropeNumber/(float)totalModels) * 2.0 + 0.1;
		// Call the single polytrope for this thread
		single_polytrope(xiL, state, nXi, polytropicIndex, parsedArgv, polytropeNumber);
	}
}

void errorCheck(int code, hipError_t err)
{
    if(err != hipSuccess) {
        printf("%s in %s at line %d (ERR NUM %d)\n",hipGetErrorString(err),__FILE__,__LINE__,code);
        exit(EXIT_FAILURE);
	}
}

double* int_n_model(double* xiL_H, double xi0, double xif, double h, int models, long int nXi, double* parsedArgv, int argc){
	double* oList; // Output List - To be filled
	double* xiList;// xi list
	double* pargv; // command line argument list

	// Allocate and Copy Data from host to device
	errorCheck(1, hipMalloc((void **) &xiList, sizeof(double)*(((xif-xi0)/h)+1)));
	errorCheck(2, hipMemcpy(xiList, xiL_H, sizeof(double)*(((xif-xi0)/h)+1), hipMemcpyHostToDevice)); 
	errorCheck(3, hipMalloc((void **) &oList, sizeof(double)*nXi*2*models));	
	errorCheck(4, hipMalloc((void **) &pargv, sizeof(double)*(argc-1)));
	errorCheck(5, hipMemcpy(pargv, parsedArgv, sizeof(double)*(argc-1), hipMemcpyHostToDevice));	

	// Base the CUDA grid size on the the models requested
	int TILELENGTH = 10;
	dim3 dimGrid(ceil(models/(float)TILELENGTH), 1, 1); dim3 dimBlock(TILELENGTH, 1, 1);

	distribute_jobs<<<dimGrid, dimBlock>>>(xiList, oList, nXi, models, TILELENGTH, pargv);
	// Wait for all threads to complete
	hipDeviceSynchronize();

	double* state = new double[2*nXi*models];
	// Copy data from device to host
	errorCheck(6, hipMemcpy(state, oList, sizeof(double)*2*nXi*models, hipMemcpyDeviceToHost));
	return state;
}

__device__ double rk4(double yN, float h, odeModel model, double *argv, int argc){
	double k1, k2, k3, k4;
	k1 = h*model(yN, argv, argc);
	k2 = h*model(yN+k1/2, argv, argc);
	k3 = h*model(yN+k2/2, argv, argc);
	k4 = h*model(yN+k3, argv, argc);

	return yN+(k1/6.0)+(k2/3.0)+(k3/3.0)+(k4/6.0);
}
