#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>

using namespace std;

__device__ void single_polytrope(double* state, long int nXi, int totalModels, int TILELENGTH){
	__shared__ double modelArgv[3];
	polytropeNumber = blockIdx.x*TILELENGTH + threadIdx.x;

	if (polytropeNumber < totalModels)
	{
		statePtr = state + sizeof(double)*nXi*2*polytropeNumber;
		for(int i=0; i<nXi; i++){
			if (i==0){
				// Set the initial theta(xi) value based on the power serise expansion
				statePtr[1][i] = parsedArgv[0];
			}	
			else{
				modelArgv[0] = statePtr[0][i];
				modelArgv[1] = statePtr[1][i-1];
				// Set up a window which is equivilent to zero to prevent the integrator
				// from jumping over zero
				if (statePtr[1][i-1] > 1.0e-5){
					// Integrate with rk4
					statePtr[2][i] = rk4(state[2][i-1], h, (odeModel)vdot_degenerate, modelArgv, 2);
					statePtr[1][i] = state[2][i]*h + state[1][i-1];
				}
				// When the dimensionless density goes negative constrain it to zero
				else{
					statePtr[1][i] = 0;
					statePtr[2][i] = 0;
				}
			}
			// Only keep track of the mass where the equation is defined
			if (statePtr[1][i] > 0){
				// Left endpoint reiemann-sum
				dm = pow(statePtr[0][i], 2)*state[1][i]*h;
				m += dm;
			}
		}
	}
}

void errorCheck(int code, hipError_t err)
{
    if(err != hipSuccess) {
        printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }
}

void int_n_model(double* xiL_H, double xi0, double xif, double h, int models){
	double* oList;
	double* xiList;
	errorCheck(1, hipMalloc((void **) &xiList, sizeof(double)*(((xif-xi0)/h)+1)));
	errorCheck(2, hipMemcpy(xiList, xiL_H, sizeof(double)*(((xif-xi0)/h)+1), hipMemcpyHostToDevice)); 
	errorCheck(3, hipMalloc((void **) &oList, sizeof(double)*nXi*2*models));	



}
