#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/complex.h>

#include "model.cuh"


__device__ double a(int k, float n){
	// Recursion halting conditions based on boundary conditions
	// that theta(0) = 1 and that thetadot(0) = 0
	if (k == 0){
		return 1;
	}
	else if (k == 1){
		return 0;
	}
	else{
		// relation determined between a, c, and k in part one of project
		return -(c(k-2, n)/(pow((double)k, (double)2)+k));
	}
}


__device__ double c(int m, float n){
	// halting condition from formal power serise definition
	if (m == 0){
		return pow((double)a(0, n), (double)n);
	}
	else{
		double sum = 0;
		for (int k = 1; k <= m; k++){
			// relation from formal power serise
			sum += (k*n-m+k)*a(k, n)*c(m-k, n);
		}
		return (1/(m*a(0, n)))*sum;
	}

}


__device__ double theta_approx(double xi, float n, int itr){
	double theta = 0;
	for (int k=0; k<itr; k++){
		// General form of a power serise
		theta += a(k, n)*pow((double)xi, (double)k);
	}
	return theta;
}
